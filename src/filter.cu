#include "hip/hip_runtime.h"
#include <cutil.h>
#include "filter.h"
#include "timer.h"
#include "effects.h"
#include "symbol.h"
#include "image_util.h"
#include "blue_noise.h"
#include "bspline3_sampler.h"
#include "recfilter.h"

#define USE_LAUNCH_BOUNDS 1
const int BW_F1 = 32, // cuda block width
#if SAMPDIM == 8
          BH_F1 = 8; 
#else
          BH_F1 = 16;
#endif

const int BW_F2 = 32,
          BH_F2 = 8; 

#if USE_LAUNCH_BOUNDS
const int 
#if SAMPDIM == 8 && CUDA_SM >= 20
          NB_F1 = 2,  // number of blocks resident per SM
#else
          NB_F1 = 1,  // number of blocks resident per SM
#endif
          NB_F2 = 4;
#endif

__constant__ float2 blue_noise[SAMPDIM];
__constant__ float bspline3_data[SAMPDIM*KS*KS];

__constant__ filter_operation filter_op;

// do the actual value processing according to what's in 'filter_op'
template <effect_type OP, class S>
__device__ typename S::result_type do_filter(const S &sampler, float2 pos)
{
    switch(OP)
    {
    case EFFECT_POSTERIZE:
        return posterize(sampler(pos), filter_op.levels);
    case EFFECT_SCALE:
        return scale(sampler(pos),filter_op.scale);
    case EFFECT_BIAS:
        return bias(sampler(pos),filter_op.bias);
    case EFFECT_ROOT:
        return root(sampler(pos),filter_op.degree);
    case EFFECT_THRESHOLD:
        return threshold(sampler(pos),filter_op.threshold);
    case EFFECT_REPLACEMENT:
        return replacement(sampler(pos), 
                           filter_op.old_color, 
                           filter_op.new_color, 
                           filter_op.tau);
    case EFFECT_GRADIENT_EDGE_DETECTION:
        return gradient_edge_detection(sampler(pos,1,0),sampler(pos,0,1));
    case EFFECT_LAPLACIAN:
        return laplacian(sampler(pos,2,0),sampler(pos,0,2));
    case EFFECT_LAPLACE_EDGE_ENHANCEMENT:
        return laplace_edge_enhancement(sampler(pos),
                                        sampler(pos,2,0),sampler(pos,0,2),
                                        filter_op.multiple);
    case EFFECT_YAROSLAVSKY_BILATERAL:
        return yaroslavsky_bilateral(sampler(pos),
                                     sampler(pos,1,0), sampler(pos,0,1),
                                     sampler(pos,1,1),
                                     sampler(pos,2,0),sampler(pos,0,2),
                                     filter_op.rho, filter_op.h);
    case EFFECT_BRIGHTNESS_CONTRAST:
        return brightness_contrast(sampler(pos),filter_op.brightness,
                                   filter_op.contrast);
    case EFFECT_HUE_SATURATION_LIGHTNESS:
        return hue_saturation_lightness(sampler(pos),filter_op.hue,
                                   filter_op.saturation,filter_op.lightness);
    case EFFECT_IDENTITY:
    default:
        return sampler(pos);
    }
}

template <int C>
struct filter_traits {};

template <int C>
struct sum_traits
    : pixel_traits<float,C+1>
{
    typedef typename pixel_traits<float,C+1>::pixel_type type;
};

struct filter_plan
{
    virtual ~filter_plan() {}

    int flags;
    hipArray *a_in;

    filter_operation op;

    recfilter5_plan *recfilter_plan;
};

template <int C>
struct filter_plan_C : filter_plan
{
    dimage<typename sum_traits<C>::type,KS*KS> temp_image;
};

void init_blue_noise()/*{{{*/
{
    std::vector<float2> blue_noise;
    std::vector<float> bspline3_data;
    blue_noise.reserve(SAMPDIM);
    bspline3_data.reserve(SAMPDIM*KS*KS);
    for(int i=0; i<SAMPDIM; ++i)
    {
        float2 n = make_float2(blue_noise_x[i], blue_noise_y[i]);

        blue_noise.push_back(n);
        for(int y=0; y<KS; ++y)
        {
            for(int x=0; x<KS; ++x)
            {
                bspline3_data.push_back(bspline3(x+n.x-1.5)*
                                        bspline3(y+n.y-1.5)/SAMPDIM);
            }
        }
    }
    copy_to_symbol("blue_noise",blue_noise);
    copy_to_symbol("bspline3_data",bspline3_data);
}/*}}}*/

template<int C>
void copy_to_array(hipArray *out, dimage_ptr<const float,C> in);

template<int C> 
filter_plan *
filter_create_plan(dimage_ptr<const float,C> img, const filter_operation &op,/*{{{*/
            int flags)
{
    assert(!img.empty());

    typedef filter_traits<C> cfg;
    typedef typename pixel_traits<float,C>::texel_type texel_type;
    typedef typename sum_traits<C>::type sum_type;

    filter_plan_C<C> *plan = new filter_plan_C<C>;

    plan->flags = flags;
    plan->op = op;

    int imgsize = img.width()*img.height();

    Vector<float,1+1> weights;

    // calculate cubic b-spline weights
    float a = 2.f-std::sqrt(3.0f);

    weights[0] = 1+a;
    weights[1] = a;

    if(op.post_filter == FILTER_CARDINAL_BSPLINE3 ||
       op.pre_filter == FILTER_CARDINAL_BSPLINE3)
    {
        plan->recfilter_plan = 
            recfilter5_create_plan<1>(img.width(),img.height(),img.rowstride(),
                                      weights, CLAMP_TO_EDGE, 1);
    }
    else
        plan->recfilter_plan = NULL;

    base_timer *timer = NULL;

    // copy the input data to a texture
    hipChannelFormatDesc ccd = hipCreateChannelDesc<texel_type>();

    hipMallocArray(&plan->a_in, &ccd, img.width(),img.height());

    if(op.post_filter == FILTER_CARDINAL_BSPLINE3)
    {
        dimage<float,C> preproc_img(img.width(), img.height());

        if(flags & VERBOSE)
            timer = &timers.gpu_add("Convolve with bspline3^-1",
                                    img.width()*img.height(), "P");

        // convolve with a bpsline3^-1 to make a cardinal post-filter
        for(int i=0; i<C; ++i)
            recfilter5(plan->recfilter_plan, preproc_img[i], img[i]);

        if(timer)
            timer->stop();

        copy_to_array(plan->a_in, dimage_ptr<const float,C>(&preproc_img));
    }
    else
        copy_to_array(plan->a_in, img);

    cfg::tex().normalized = false;
    cfg::tex().filterMode = hipFilterModeLinear;

    cfg::tex().addressMode[0] = cfg::tex().addressMode[1] = hipAddressModeClamp;

    copy_to_symbol("filter_op",op);

    plan->temp_image.resize(img.width(), img.height());

    init_blue_noise();

    return plan;
}/*}}}*/

void filter_free(filter_plan *plan)/*{{{*/
{
    if(plan == NULL)
        return;

    hipFreeArray(plan->a_in);
    delete plan;

    recfilter5_free(plan->recfilter_plan);
}/*}}}*/

template <effect_type OP,int C>
__global__
#if USE_LAUNCH_BOUNDS
__launch_bounds__(BW_F1*BH_F1, NB_F1)
#endif
void filter_kernel1(dimage_ptr<typename sum_traits<C>::type,KS*KS> out)/*{{{*/
{
    int tx = threadIdx.x, ty = threadIdx.y;

    int x = blockIdx.x*BW_F1+tx, y = blockIdx.y*BH_F1+ty;

    if(!out.is_inside(x,y))
        return;

    // output will point to the pixel we're processing now
    int idx = out.offset_at(x,y);
    out += idx;

    // we're using some smem as registers not to blow up the register space,
    // here we define how much 'registers' are in smem, the rest is used
    // in regular registers
    
    typedef filter_traits<C> cfg;

    typedef typename sum_traits<C>::type sum_type;
    typedef typename pixel_traits<float,C>::pixel_type pixel_type;

    const int SMEM_SIZE = cfg::smem_size,
              REG_SIZE = KS*KS-SMEM_SIZE;

    __shared__ sum_type _sum[BH_F1][SMEM_SIZE][BW_F1];
    sum_type (*ssum)[BW_F1] = (sum_type (*)[BW_F1]) &_sum[ty][0][tx];

    sum_type sum[REG_SIZE];

    // Init registers to zero
    for(int i=0; i<REG_SIZE; ++i)
        sum[i] = sum_traits<C>::make_pixel(0);

#pragma unroll
    for(int i=0; i<SMEM_SIZE; ++i)
        *ssum[i] = sum_traits<C>::make_pixel(0);

    // top-left position of the kernel support
    float2 p = make_float2(x,y)-1.5f+0.5f;

    float *bspline3 = bspline3_data;

    bspline3_sampler<typename cfg::texfetch_type> sampler;

    for(int s=0; s<SAMPDIM; ++s)
    {
        pixel_type value = do_filter<OP>(sampler, p+blue_noise[s]);
        value = srgb2lrgb(value);

        // scans through the kernel support, collecting data for each position
#pragma unroll
        for(int i=0; i<SMEM_SIZE; ++i)
        {
            float wij = bspline3[i];

            *ssum[i] += sum_traits<C>::make_pixel(value*wij, wij);
        }
        bspline3 += SMEM_SIZE;
#pragma unroll
        for(int i=0; i<REG_SIZE; ++i)
        {
            float wij = bspline3[i];

            sum[i] += sum_traits<C>::make_pixel(value*wij, wij);
        }
        bspline3 += REG_SIZE;
    }

    // writes out to gmem what's in the registers
#pragma unroll
    for(int i=0; i<SMEM_SIZE; ++i)
        *out[i] = *ssum[i];

#pragma unroll
    for(int i=0; i<REG_SIZE; ++i)
        *out[SMEM_SIZE+i] = sum[i];
}/*}}}*/

template <int C>
__global__
#if USE_LAUNCH_BOUNDS
__launch_bounds__(BW_F2*BH_F2, NB_F2)
#endif
void filter_kernel2(dimage_ptr<float,C> out, /*{{{*/
                    dimage_ptr<const typename sum_traits<C>::type,KS*KS> in)
{
    int tx = threadIdx.x, ty = threadIdx.y;

    int x = blockIdx.x*BW_F2+tx, y = blockIdx.y*BH_F2+ty;

    // out of bounds? goodbye
    if(!in.is_inside(x,y))
        return;

    // in and out points to the input/output pixel we're processing
    int idx = in.offset_at(x,y);
    in += idx;
    out += idx;

    // treat corner cases where the support is outside the image
    int mi = min(y+KS,in.height())-y,
        mj = min(x+KS,in.width())-x;

    // sum the contribution of nearby pixels
    typename sum_traits<C>::type sum = sum_traits<C>::make_pixel(0);

#pragma unroll
    for(int i=0; i<mi; ++i)
    {
#pragma unroll
        for(int j=0; j<mj; ++j)
        {
            sum += *in[i*KS+j];
            ++in;
        }
        in += in.rowstride()-mj;
    }

    *out = filter_traits<C>::normalize_sum(sum);
}/*}}}*/

template <int C>
void filter(filter_plan *_plan, dimage_ptr<float,C> out, const filter_operation &op)/*{{{*/
{
    filter_plan_C<C> *plan = dynamic_cast<filter_plan_C<C> *>(_plan);
    assert(plan != NULL);

    if(plan->op.post_filter != op.post_filter)
        throw std::runtime_error("Postfilter changed, plan must be recreated");

    copy_to_symbol("filter_op",op);

    typedef filter_traits<C> cfg;
    assert(plan->temp_image.width() == out.width() &&
           plan->temp_image.height() == out.height());

    hipBindTextureToArray(cfg::tex(), plan->a_in);

    dim3 bdim(BW_F1,BH_F1),
         gdim((out.width()+bdim.x-1)/bdim.x, (out.height()+bdim.y-1)/bdim.y);

    base_timer *timer = NULL;

    if(plan->flags & VERBOSE)
        timer = &timers.gpu_add("First pass",out.width()*out.height(),"P");


#define CASE(EFFECT) \
    case EFFECT:\
        filter_kernel1<EFFECT,C><<<gdim, bdim>>>(&plan->temp_image); \
        break

    switch(op.type)
    {
    CASE(EFFECT_IDENTITY);
    CASE(EFFECT_POSTERIZE);
    CASE(EFFECT_SCALE);
    CASE(EFFECT_BIAS);
    CASE(EFFECT_ROOT);
    CASE(EFFECT_THRESHOLD);
    CASE(EFFECT_REPLACEMENT);
    CASE(EFFECT_GRADIENT_EDGE_DETECTION);
    CASE(EFFECT_LAPLACIAN);
    CASE(EFFECT_LAPLACE_EDGE_ENHANCEMENT);
    CASE(EFFECT_YAROSLAVSKY_BILATERAL);
    CASE(EFFECT_BRIGHTNESS_CONTRAST);
    CASE(EFFECT_HUE_SATURATION_LIGHTNESS);
    default:
        assert(false);
    }
#undef CASE

    if(timer)
        timer->stop();
                   
    {
        if(plan->flags & VERBOSE)
            timer = &timers.gpu_add("Second pass",out.width()*out.height(),"P");

        dim3 bdim(BW_F2,BH_F2),
             gdim((out.width()+bdim.x-1)/bdim.x,(out.height()+bdim.y-1)/bdim.y);
        filter_kernel2<C><<<gdim, bdim>>>(out, &plan->temp_image);

        if(timer)
            timer->stop();
    }

    hipUnbindTexture(cfg::tex());

    if(op.pre_filter == FILTER_CARDINAL_BSPLINE3)
    {
        if(plan->flags & VERBOSE)
            timer = &timers.gpu_add("Convolve with bspline3^-1",out.width()*out.height(),"P");

        // convolve with a bpsline3^-1 to make a cardinal pre-filter
        for(int i=0; i<C; ++i)
            recfilter5(plan->recfilter_plan, out[i]);

        if(timer)
            timer->stop();
    }

    // maps back to gamma space
    lrgb2srgb(out, out);
}/*}}}*/

// Grayscale filtering ===================================================/*{{{*/

texture<float, 2, hipReadModeElementType> t_in_gray;

struct texfetch_gray
{
    typedef float result_type;

    __device__ float operator()(float x, float y)
    {
        return tex2D(t_in_gray, x, y);
    }
};

template <> 
struct filter_traits<1>
{
    typedef texfetch_gray texfetch_type;
    static const int smem_size = 3;

    static 
    texture<float,2,hipReadModeElementType> &tex() { return t_in_gray; }

    __device__ static float normalize_sum(float2 sum)
    {
        return sum.x / sum.y;
    }
};

template<>
void copy_to_array(hipArray *out, dimage_ptr<const float> in)
{
    hipMemcpy2DToArray(out, 0, 0, in, 
                        in.rowstride()*sizeof(float),
                        in.width()*sizeof(float), in.height(),
                        hipMemcpyDeviceToDevice);
}

template 
void filter(filter_plan *, dimage_ptr<float,1> img, const filter_operation &op);

template 
filter_plan *
filter_create_plan(dimage_ptr<const float,1> img, const filter_operation &op, 
                   int flags);
/*}}}*/

//{{{ RGB filtering =========================================================

texture<float4, 2, hipReadModeElementType> t_in_rgba;

struct texfetch_rgba
{
    typedef float3 result_type;

    __device__ float3 operator()(float x, float y)
    {
        return make_float3(tex2D(t_in_rgba, x, y));
    }
};

template <> 
struct filter_traits<3>
{
    typedef texfetch_rgba texfetch_type;

#if CUDA_SM >= 20
    static const int smem_size = 5;
#else
    static const int smem_size = 3;
#endif

    static int flags;

    static texture<float4,2,hipReadModeElementType> &tex() 
        { return t_in_rgba; }

    __device__ static float3 normalize_sum(float4 sum)
    {
        return make_float3(sum) / sum.w;
    }
};

template <>
void copy_to_array(hipArray *out, dimage_ptr<const float,3> img)
{
    dimage<float3> temp;
    temp.resize(img.width(), img.height());
    convert(&temp, img);

    hipMemcpy2DToArray(out, 0, 0, temp, 
                        temp.rowstride()*sizeof(float4),
                        temp.width()*sizeof(float4), temp.height(),
                        hipMemcpyDeviceToDevice);
}

template 
void filter(filter_plan *, dimage_ptr<float,3> img, const filter_operation &op);

template 
filter_plan *
filter_create_plan(dimage_ptr<const float,3> img, const filter_operation &op, 
                   int flags);
/*}}}*/
