#include "hip/hip_runtime.h"
// Copyright 2010--2020 Andre Maximo, Rodolfo Schulz de Lima, Diego Nehab
//
// Licensed under the Apache License, Version 2.0 (the "License"); you may not
// use this file except in compliance with the License. You may obtain a copy of
// the License at http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
// WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
// License for the specific language governing permissions and limitations under
// the License.

//== INCLUDES =================================================================

#include <cmath>
#include <cfloat>
#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <iostream>
#include <algorithm>
#include <complex>
#include "config.h"

#include "symbol.h"
#include "dvector.h"
#include "util.h"

#include "recfilter.h"
#include "image.h"

#if CUDA_SM >= 20
#   define W1 8
#   define NB1 6

#   define W23 8
#   define NB23 6

#   define W45 7
#   define NB45 5

#   define W6 8
#   define NB6 7
#else
#   define W1 8
#   define NB1 4

#   define W23 8
#   define NB23 0

#   define W45 8
#   define NB45 0

#   define W6 8
#   define NB6 4
#endif

/*! @mainpage recursive-filtering

\section notes Notes

Naming conventions are: c_ constant; t_ texture; g_ global memory; s_
shared memory; d_ device pointer; a_ cuda-array; p_ template
parameter; f_ surface.

*/

#define PREFIX(x) PP_CAT3(c5_, ORDER,_##x)

#define c_weights PREFIX(weights)
#define c_AbF_T PREFIX(AbF_T)
#define c_AbR_T PREFIX(AbR_T)
#define c_HARB_AFP_T PREFIX(HARB_AFP_T)
#define c_AbF PREFIX(AbF)
#define c_AbR PREFIX(AbR)
#define c_HARB_AFP PREFIX(HARB_AFP)
#define c_ARE_T PREFIX(ARE_T)
#define c_HARB_AFB PREFIX(HARB_AFB)
#define c_TAFB PREFIX(TAFB)
#define c_ARB_AFP_T PREFIX(ARB_AFP_T)

#ifndef NON_ORDER_DEPENDENT_PARAMETERS_DEFINED
#define NON_ORDER_DEPENDENT_PARAMETERS_DEFINED 1
__constant__ int c_width, c_height, c_rowstride,
                 c_adj_width, c_adj_height,
                 c_m_size, // number of column-blocks,
                 c_n_size, // number of row-blocks,
                 c_last_m, c_last_n,
                 c_border;
__constant__ float c_inv_width, c_inv_height;
#endif

__constant__ Vector<float,ORDER+1> c_weights;

__constant__ Matrix<float,ORDER,ORDER> c_AbF_T, c_AbR_T, 
                                       c_HARB_AFP_T,
                               c_AbF, c_AbR, c_HARB_AFP;
__constant__ Matrix<float,ORDER,WS> c_ARE_T, c_HARB_AFB, 
                                    c_TAFB, c_ARB_AFP_T;


#ifndef TEXTURE_DEFINED
#define TEXTURE_DEFINED 1
texture<float, hipTextureType2D, hipReadModeElementType> t_in;
#endif

//=== IMPLEMENTATION ==========================================================

#ifndef AUX_FUNCS_DEFINED
#define AUX_FUNCS_DEFINED 1

template <int W, int U, int V>
__device__
void read_block(Matrix<float,U,V> &block, int m, int n, 
                float inv_width, float inv_height)
{
    int tx = threadIdx.x, ty = threadIdx.y;

    float tu = ((m-c_border)*WS+tx+.5f)*inv_width,
          tv = ((n-c_border)*WS+ty+.5f)*inv_height;

    float (*bdata)[V] = (float (*)[V]) &block[ty][tx]
#if CUDA_SM >= 20
          ,(*bdata2)[V] = (float (*)[V])&block[ty+WS][tx]
#endif
          ;

#pragma unroll
    for(int i=0; i<WS-(WS%W); i+=W)
    {
        **bdata = tex2D(t_in, tu, tv);
        bdata += W;

#if CUDA_SM >= 20
        **bdata2 = tex2D(t_in, tu+WS*inv_width, tv);
        bdata2 += W;
#endif

        tv += W*inv_height;
    }

    if(ty < WS%W)
    {
        **bdata = tex2D(t_in, tu, tv);
#if CUDA_SM >= 20
        **bdata2 = tex2D(t_in, tu+WS*inv_width, tv);
#endif
    }
}

template <int W, int U, int V>
__device__
void write_block(float *out,
                 const Matrix<float,U,V> &block, 
                 int width, int height, int rowstride,
                 int m, int n, int last_m, int last_n)
{
    int tx = threadIdx.x, ty = threadIdx.y;

    // current block intersects transp_out's area?
//    if(m >= c_border && m <= last_m && n >= c_border && n <= last_n)
    {
        int y = (n-c_border)*WS, 
            x = (m-c_border)*WS+tx;

        out += y*rowstride + x;

  //      if(y < height)
        {
            int maxy = min(height, y+WS);
            for(int i=0; y<maxy; ++y, ++i, out += width)
                *out = block[ty*WS+i][tx];

        }
    }
}

template <class T, int R>
__device__ 
Vector<T,R> mad(Matrix<T,R,WS> &r, const Vector<T,R> &a, 
                    const Matrix<T,R,R> &b)
{
#pragma unroll
    for(int j=0; j<R; ++j)
    {
        T acc = *r[j];
#pragma unroll
        for(int i=0; i<R; ++i)
            acc += a[i]*b[i][j];
        *r[j] = acc;
    }

    return r.col(0);
}

template <class T, int R>
__device__ 
Vector<T,R> mad(Matrix<T,R,WS> &r, const Matrix<T,R,R> &a,
                const Vector<T,R> &b)
{
#pragma unroll
    for(int i=0; i<R; ++i)
    {
        T acc = *r[i];
#pragma unroll
        for(int j=0; j<R; ++j)
            acc += a[i][j]*b[j];
        *r[i] = acc;
    }

    return r.col(0);
}

template <class T, int R>
__device__ 
void mad(Matrix<T,R,WS> &r, const Matrix<T,R,WS> &a, 
         const Matrix<T,R,R> &b)
{
#pragma unroll
    for(int j=0; j<R; ++j)
    {
        T acc = *r[j];
#pragma unroll
        for(int i=0; i<R; ++i)
            acc += *a[i]*b[i][j];
        *r[j] = acc;
    }
}

template <class T, int R>
__device__ 
void mad(Matrix<T,R,WS> &r, const Matrix<T,R,R> &a, const Matrix<T,R,WS> &b)
{
#pragma unroll
    for(int i=0; i<R; ++i)
    {
        T acc = *r[i];
#pragma unroll
        for(int j=0; j<R; ++j)
            acc += a[i][j]* *b[j];
        *r[i] = acc;
    }
}

template <class T, int R>
__device__ 
void mad(Matrix<T,R,WS> &r,  const Matrix<T,R,WS> &a, 
         const Matrix<T,R,WS> &b, const Matrix<T,R,WS> &c,
	    volatile T (*block_RD)[WS/2+WS+1])
{
    int tx = threadIdx.x, ty = threadIdx.y;

    Matrix<T,R,R> rint;

    for(int i=0; i<R; ++i)
    {
        for(int j=0; j<R; ++j)
        {
            block_RD[ty][tx] = a[i][tx] * *b[j];
            block_RD[ty][tx] += block_RD[ty][tx-1];
            block_RD[ty][tx] += block_RD[ty][tx-2];
            block_RD[ty][tx] += block_RD[ty][tx-4];
            block_RD[ty][tx] += block_RD[ty][tx-8];
            block_RD[ty][tx] += block_RD[ty][tx-16];
            rint[i][j] = block_RD[ty][WS-1];
        }
    }

    mad(r, rint, (const Matrix<T,R,WS> &)c[0][tx]);
}

#endif

/**
 *  @brief Algorithm 5 stage 1
 *
 *  This function computes the algorithm stage 5.1 following:
 *
 *  In parallel for all $m$ and $n$, compute and store each
 *  $P_{m,n}(\bar{Y})$, $E_{m,n}(\hat{Z})$, $P^\T_{m,n}(\check{U})$,
 *  and $E^\T_{m,n}(\tilde{V})$.
 *
 *  @param[in] g_in Input image
 *  @param[out] g_transp_ybar All P_{m,n}(\bar{Y})
 *  @param[out] g_transp_zhat All $E_{m,n}(\hat{Z})$
 *  @param[out] g_ucheck All $P^\T_{m,n}(\check{U})$
 *  @param[out] g_vtilde All $E^\T_{m,n}(\tilde{V})$
 */
__global__
#if NB1
__launch_bounds__(WS*W1, NB1)
#endif
void collect_carries(Matrix<float,ORDER,WS> *g_pybar, 
                     Matrix<float,ORDER,WS> *g_ezhat,
                     Matrix<float,ORDER,WS> *g_ptucheck, 
                     Matrix<float,ORDER,WS> *g_etvtilde)
{
    int tx = threadIdx.x, ty = threadIdx.y, 
#if CUDA_SM >= 20
        m = blockIdx.x*2, 
#else
        m = blockIdx.x, 
#endif
        n = blockIdx.y;

    // each cuda block will work on two horizontally adjacent WSxWS input data
    // blocks, so allocate enough shared memory for these.
#if CUDA_SM >= 20
    __shared__ Matrix<float,WS*2,WS+1> block;
#else
    __shared__ Matrix<float,WS,WS+1> block;
#endif

    // load data into shared memory
    read_block<W1>(block, m, n, c_inv_width, c_inv_height);

#if CUDA_SM >= 20
    m += ty;
    if(m >= c_m_size)
        return;
#endif

    __syncthreads();

#if CUDA_SM >= 20
    if(ty < 2)
#else
    if(ty == 0)
#endif
    {
        Matrix<float,ORDER,WS> 
            &pybar = (Matrix<float,ORDER,WS>&)g_pybar[n*c_m_size+m][0][tx],
            &ezhat = (Matrix<float,ORDER,WS>&)g_ezhat[n*c_m_size+m][0][tx],
            &ptucheck = (Matrix<float,ORDER,WS>&)g_ptucheck[n*c_m_size+m][0][tx],
            &etvtilde = (Matrix<float,ORDER,WS>&)g_etvtilde[n*c_m_size+m][0][tx];

        const float B0_1 = c_weights[0], B0_2 = B0_1*B0_1,
                    B0_3 = B0_2*B0_1, B0_4 = B0_2*B0_2;
        {
            float *bdata = block[tx+ty*WS];

            // calculate pybar, scan left -> right
            {
                Vector<float,ORDER> p = zeros<float,ORDER>();

                p[ORDER-1] = *bdata++;

#pragma unroll
                for(int j=1; j<WS; ++j, ++bdata)
                {
#if CUDA_SM >= 20 || ORDER>1
                    *bdata = fwd(p, *bdata, c_weights);
#else
                    *bdata = p[0] = rec_op(*bdata, p[0]*c_weights[1]);
#endif
                }

                if(m < c_m_size-1)
                    pybar.set_col(0, p*B0_1);
            }

            {
                --bdata;

                Vector<float,ORDER> e = zeros<float,ORDER>();

                e[0] = *bdata--;

#pragma unroll
                for(int j=WS-2; j>=0; --j, --bdata)
                {
#if CUDA_SM >= 20 || ORDER>1
                    *bdata = rev(*bdata, e, c_weights);
#else
                    *bdata = e[0] = rec_op(*bdata, e[0]*c_weights[1]);
#endif
                }

                if(m > 0)
                    ezhat.set_col(0, e*B0_2);
            }
        }

        {
            float (*bdata)[WS+1] = (float (*)[WS+1]) &block[ty*WS][tx];
            {
                Vector<float,ORDER> p = zeros<float,ORDER>();

                p[ORDER-1] = **bdata++;

#pragma unroll
                for(int i=1; i<WS; ++i, ++bdata)
                {
#if CUDA_SM >= 20 || ORDER>1
                    **bdata = fwd(p, **bdata, c_weights);
#else
                    **bdata = p[0] = rec_op(**bdata, p[0]*c_weights[1]);
#endif
                }

                if(n < c_n_size-1)
                    ptucheck.set_col(0, p*B0_3);
            }

            if(n > 0)
            {
                --bdata;

                Vector<float,ORDER> e = zeros<float,ORDER>();

                e[0] = **bdata--;

#pragma unroll
                for(int i=WS-2; i>=0; --i, --bdata)
                {
#if CUDA_SM >= 20 || ORDER>1
                    rev(**bdata, e, c_weights);
#else
                    e[0] = rec_op(**bdata, e[0]*c_weights[1]);
#endif
                }

                etvtilde.set_col(0, e*B0_4);
            }
        }
    }
}

/**
 *  @brief Algorithm 4 stage 2 and 3 (fusioned)
 *
 *  This function computes the algorithm stages 5.2 and 5.3 following:
 *
 *  In parallel for all $n$, sequentially for each $m$, compute and
 *  store the $P_{m,n}(Y)$ according to (37) and using the previously
 *  computed $P_{m-1,n}(\bar{Y})$.
 *
 *  with simple kernel fusioned (going thorough global memory):
 *
 *  In parallel for all $n$, sequentially for each $m$, compute and
 *  store $E_{m,n}(Z)$ according to (45) using the previously computed
 *  $P_{m-1,n}(Y)$ and $E_{m+1,n}(\hat{Z})$.
 *
 *  @param[in,out] g_transp_ybar All $P_{m,n}(\bar{Y})$
 *  @param[in,out] g_transp_zhat All $E_{m,n}(\hat{Z})$
 */
__global__
#if NB23
__launch_bounds__(WS*W23, NB23)
#endif
void adjust_carries(Matrix<float,ORDER,WS> *g_pybar, 
                    Matrix<float,ORDER,WS> *g_ezhat,
                    int m_size, int n_size)
{
    int tx = threadIdx.x, ty = threadIdx.y, n = blockIdx.y;

    __shared__ Matrix<float,ORDER,WS> block[W23];

    Matrix<float,ORDER,WS> &bdata = (Matrix<float,ORDER,WS> &)block[ty][0][tx];

    // P(ybar) -> P(y) processing --------------------------------------

    Matrix<float,ORDER,WS> *pybar = (Matrix<float,ORDER,WS> *)&g_pybar[n*m_size+ty][0][tx];

    Vector<float,ORDER> py = zeros<float,ORDER>(); // P(Y)

    int m = 0;
    if(blockDim.y == W23)
    {
        int mmax = m_size-(m_size%W23)-1;
        for(; m<mmax; m+=W23)
        {
            // read P(Y)
            bdata.set_col(0, pybar->col(0));

            __syncthreads();

            if(ty == 0)
            {
                Matrix<float,ORDER,WS> *bdata = (Matrix<float,ORDER,WS> *)&block[0][0][tx];
#pragma unroll
                for(int dm=0; dm<W23; ++dm, ++bdata)
                    py = mad(bdata[0], py, c_AbF_T);
            }

            __syncthreads();

            pybar->set_col(0,bdata.col(0));

            pybar += W23;
        }
    }

    // remaining column-blocks

    if(m < m_size-1)
    {
        if(m+ty < m_size-1)
            bdata.set_col(0, pybar->col(0));

        int remaining = m_size-1 - m;

        __syncthreads();

        if(ty == 0)
        {
            Matrix<float,ORDER,WS> *bdata = (Matrix<float,ORDER,WS> *)&block[0][0][tx];
#pragma unroll
            for(int dm=0; dm<remaining; ++dm, ++bdata)
                py = mad(bdata[0], py, c_AbF_T);
        }

        __syncthreads();

        if(m+ty < m_size-1)
            pybar->set_col(0,bdata.col(0));
    }


    // E(zhat) -> E(z) processing --------------------------------------

    m = m_size-1;

    Matrix<float,ORDER,WS> 
        *pm1y  = (Matrix<float,ORDER,WS> *)&g_pybar[n*m_size+m-ty-1][0][tx],
        *ezhat = (Matrix<float,ORDER,WS> *)&g_ezhat[n*m_size+m-ty][0][tx];


    // all pybars must be updated!
    __syncthreads();

    Vector<float,ORDER> ez = zeros<float,ORDER>();

    m = m_size-1;
    if(blockDim.y == W23)
    {
        int mmin = m_size%W23;
        for(; m>=mmin; m-=W23)
        {
            if(m > 0)
            {
                bdata.set_col(0, ezhat->col(0));

                if(m-ty > 0)
                    mad(bdata, *pm1y, c_HARB_AFP_T);

                __syncthreads();

                if(ty == 0)
                {
                    Matrix<float,ORDER,WS> *bdata 
                        = (Matrix<float,ORDER,WS> *)&block[0][0][tx];
#pragma unroll
                    for(int dm=0; dm<W23; ++dm, ++bdata)
                        ez = mad(bdata[0], ez, c_AbR_T);
                }

                __syncthreads();

                ezhat->set_col(0,bdata.col(0));
            }

            ezhat -= W23;
            pm1y -= W23;
        }
    }

    // remaining column-blocks

    if(m > 0)
    {
        int remaining = m+1;

        if(m-ty > 0)
        {
            bdata.set_col(0, ezhat->col(0));
            mad(bdata, *pm1y, c_HARB_AFP_T);
        }

        __syncthreads();

        if(ty == 0)
        {
            Matrix<float,ORDER,WS> *bdata = (Matrix<float,ORDER,WS> *)&block[0][0][tx];
#pragma unroll
            for(int dm=1; dm<remaining; ++dm, ++bdata)
                ez = mad(bdata[0], ez, c_AbR_T);
        }

        __syncthreads();

        if(m-ty > 0)
            ezhat->set_col(0,bdata.col(0));
    }
}

/**
 *  @brief Algorithm 5 stage 4 and 5 (fusioned)
 *
 *  This function computes the algorithm stages 5.2 and 5.3 following:
 *
 *  In parallel for all $n$, sequentially for each $m$, compute and
 *  store the $P_{m,n}(Y)$ according to (37) and using the previously
 *  computed $P_{m-1,n}(\bar{Y})$.
 *
 *  with simple kernel fusioned (going thorough global memory):
 *
 *  In parallel for all $n$, sequentially for each $m$, compute and
 *  store $E_{m,n}(Z)$ according to (45) using the previously computed
 *  $P_{m-1,n}(Y)$ and $E_{m+1,n}(\hat{Z})$.
 *
 *  @param[in,out] g_transp_ybar All $P_{m,n}(\bar{Y})$
 *  @param[in,out] g_transp_zhat All $E_{m,n}(\hat{Z})$
 */
__global__
#if NB45
__launch_bounds__(WS*W45, NB45)
#endif
void adjust_carries(Matrix<float,ORDER,WS> *g_ptucheck, 
                    Matrix<float,ORDER,WS> *g_etvtilde,
                    Matrix<float,ORDER,WS> *g_py, 
                    Matrix<float,ORDER,WS> *g_ez,

                    int m_size, int n_size)
{
    int tx = threadIdx.x, ty = threadIdx.y, m = blockIdx.x;

    __shared__ Matrix<float,ORDER,WS> block[W45];

	volatile __shared__ float block_RD_raw[W45][WS/2+WS+1];
	volatile float (*block_RD)[WS/2+WS+1] = 
            (float (*)[WS/2+WS+1]) &block_RD_raw[0][WS/2];
    if(ty < W45)
        block_RD_raw[ty][tx] = 0;

    Matrix<float,ORDER,WS> &bdata = (Matrix<float,ORDER,WS> &)block[ty][0][tx];

    // Pt(ucheck) -> Pt(u) processing --------------------------------------

    Matrix<float,ORDER,WS> 
        *ptucheck = (Matrix<float,ORDER,WS> *)&g_ptucheck[ty*c_m_size+m][0][tx],
        *pm1y = (Matrix<float,ORDER,WS> *)&g_py[ty*c_m_size+m-1][0][tx],
        *em1z = (Matrix<float,ORDER,WS> *)&g_ez[ty*c_m_size+m+1][0][tx];

    Vector<float,ORDER> ptu = zeros<float,ORDER>(); // Pt(U)

    int n = 0;
    if(blockDim.y == W45)
    {
        int nmax = n_size-(n_size%W45)-1;
        for(; n<nmax; n+=W45)
        {
            // read Pt(U)
            bdata.set_col(0, ptucheck->col(0));

            if(m > 0)
                mad(bdata, c_TAFB, *pm1y, c_ARB_AFP_T, block_RD);

            if(m < c_m_size-1)
                mad(bdata, c_TAFB, *em1z, c_ARE_T, block_RD);

            __syncthreads();

            if(ty == 0)
            {
                Matrix<float,ORDER,WS> *bdata = (Matrix<float,ORDER,WS> *)&block[0][0][tx];
#pragma unroll
                for(int dn=0; dn<W45; ++dn, ++bdata)
                    ptu = mad(*bdata, c_AbF, ptu);
            }

            __syncthreads();

            ptucheck->set_col(0,bdata.col(0));

            ptucheck += W45*c_m_size;
            pm1y += W45*c_m_size;
            em1z += W45*c_m_size;
        }
    }

    // remaining column-blocks

    if(n < c_n_size-1)
    {
        if(n+ty < c_n_size-1)
        {
            bdata.set_col(0, ptucheck->col(0));

            if(m < c_m_size-1)
                mad(bdata, c_TAFB, *em1z, c_ARE_T, block_RD);

            if(m > 0)
                mad(bdata, c_TAFB, *pm1y, c_ARB_AFP_T, block_RD);
        }

        int remaining = n_size-1 - n;

        __syncthreads();

        if(ty == 0)
        {
            Matrix<float,ORDER,WS> *bdata = (Matrix<float,ORDER,WS> *)&block[0][0][tx];
#pragma unroll
            for(int dn=0; dn<remaining; ++dn, ++bdata)
                ptu = mad(bdata[0], c_AbF, ptu);
        }

        __syncthreads();

        if(n+ty < n_size-1)
            ptucheck->set_col(0,bdata.col(0));
    }

    // E(zhat) -> E(z) processing --------------------------------------

    n = n_size-1;

    Matrix<float,ORDER,WS>
        *etvtilde = (Matrix<float,ORDER,WS> *)&g_etvtilde[(n-ty)*c_m_size+m][0][tx],
        *ptn1u = (Matrix<float,ORDER,WS> *)&g_ptucheck[(n-ty-1)*c_m_size+m][0][tx];

    pm1y = (Matrix<float,ORDER,WS> *)&g_py[(n-ty)*c_m_size+m-1][0][tx];
    em1z = (Matrix<float,ORDER,WS> *)&g_ez[(n-ty)*c_m_size+m+1][0][tx];

    // all pybars must be updated!
    __syncthreads();

    Vector<float,ORDER> etv = zeros<float,ORDER>();

    if(blockDim.y == W45)
    {
        int nmin = n_size%W45;
        for(; n>=nmin; n-=W45)
        {
            if(n > 0)
            {
                bdata.set_col(0, etvtilde->col(0));

                if(m > 0)
                    mad(bdata, c_HARB_AFB, *pm1y, c_ARB_AFP_T, block_RD);

                if(m < c_m_size-1)
                    mad(bdata, c_HARB_AFB, *em1z, c_ARE_T, block_RD);

                if(n-ty > 0)
                    mad(bdata, *ptn1u, c_HARB_AFP_T);

                __syncthreads();

                if(ty == 0)
                {
                    Matrix<float,ORDER,WS> *bdata 
                        = (Matrix<float,ORDER,WS> *)&block[0][0][tx];
#pragma unroll
                    for(int dn=0; dn<W45; ++dn, ++bdata)
                        etv = mad(bdata[0], c_AbR, etv);
                }

                __syncthreads();

                etvtilde->set_col(0,bdata.col(0));
            }

            etvtilde -= W45*c_m_size;
            pm1y -= W45*c_m_size;
            em1z -= W45*c_m_size;
            ptn1u -= W45*c_m_size;
        }
    }

    // remaining column-blocks

    if(n > 0)
    {
        int remaining = n+1;

        if(n-ty > 0)
        {
            bdata.set_col(0, etvtilde->col(0));

            if(m > 0)
                mad(bdata, c_HARB_AFB, *pm1y, c_ARB_AFP_T, block_RD);

            if(m < c_m_size-1)
                mad(bdata, c_HARB_AFB, *em1z, c_ARE_T, block_RD);

            mad(bdata, *ptn1u, c_HARB_AFP_T);
        }

        __syncthreads();

        if(ty == 0)
        {
            Matrix<float,ORDER,WS> *bdata = (Matrix<float,ORDER,WS> *)&block[0][0][tx];
#pragma unroll
            for(int dn=1; dn<remaining; ++dn, ++bdata)
                etv = mad(bdata[0], c_AbR, etv);
        }

        __syncthreads();

        if(n-ty > 0)
            etvtilde->set_col(0,bdata.col(0));
    }
}

__global__
#if NB6
__launch_bounds__(WS*W6, NB6)
#endif
void write_result(float *g_out,
                  const Matrix<float,ORDER,WS> *g_py, 
                  const Matrix<float,ORDER,WS> *g_ez,
                  const Matrix<float,ORDER,WS> *g_ptu, 
                  const Matrix<float,ORDER,WS> *g_etv)
{
    int tx = threadIdx.x, ty = threadIdx.y, 
#if CUDA_SM >= 20
        m = blockIdx.x*2,
#else
        m = blockIdx.x,
#endif
        n = blockIdx.y;

    // each cuda block will work on two horizontally adjacent WSxWS input data
    // blocks, so allocate enough shared memory for these.
#if CUDA_SM >= 20
    __shared__ Matrix<float,WS*2,WS+1> block;
#else
    __shared__ Matrix<float,WS,WS+1> block;
#endif

    // load data into shared memory
    read_block<W6>(block, m, n, c_inv_width, c_inv_height);

#if CUDA_SM >= 20
    m += ty;
    if(m >= c_m_size)
        return;
#endif

    __syncthreads();

 #if CUDA_SM >= 20
    if(ty < 2)
#else
    if(ty == 0)
#endif
    {

        Matrix<float,ORDER,WS> 
            &py = (Matrix<float,ORDER,WS>&)  g_py[n*c_m_size+m-1][0][tx],
            &ez = (Matrix<float,ORDER,WS>&)  g_ez[n*c_m_size+m+1][0][tx],
            &ptu = (Matrix<float,ORDER,WS>&) g_ptu[(n-1)*c_m_size+m][0][tx],
            &etv = (Matrix<float,ORDER,WS>&) g_etv[(n+1)*c_m_size+m][0][tx];
        const float B0_2 = c_weights[0]*c_weights[0];

        {


            float *bdata = block[tx+ty*WS];

            // calculate pybar, scan left -> right
            Vector<float,ORDER> p = m==0 ? zeros<float,ORDER>()
                                     : py.col(0) / c_weights[0];

#pragma unroll
            for(int j=0; j<WS; ++j, ++bdata)
                *bdata = fwd(p, *bdata, c_weights);

            --bdata;

            Vector<float,ORDER> e = m==c_m_size-1 ? zeros<float,ORDER>()
                                              : ez.col(0);

#pragma unroll
            for(int j=WS-1; j>=0; --j, --bdata)
                *bdata = rev(*bdata*B0_2, e, c_weights);
        }
        {
            float (*bdata)[WS+1] = (float (*)[WS+1]) &block[ty*WS][tx];

            Vector<float,ORDER> p = n==0 ? zeros<float,ORDER>()
                                     : ptu.col(0) / c_weights[0];

#pragma unroll
            for(int i=0; i<WS; ++i, ++bdata)
                **bdata = fwd(p, **bdata, c_weights);

            --bdata;

            Vector<float,ORDER> e = n==c_n_size-1 ? zeros<float,ORDER>()
                                              : etv.col(0);

            // for some reason it's faster when this is here then inside the
            // next if block;
            int x = (m-c_border)*WS+tx;
            int y = (n-c_border+1)*WS-1;

            // current block intersects transp_out's area?
            if(m >= c_border && m <= c_last_m && n >= c_border && n <= c_last_n)
            {
                // image's end is in the middle of the block and we're outside
                // the image width?
                if(y >= c_height)
                {
                    // process data until we get into the image
                    int i;
#pragma unroll
                    for(i=y; i>=c_height; --i, --bdata)
                        rev(**bdata*B0_2, e, c_weights);

//                    bdata -= y-c_height+1;

                    // now we're inside the image, we must write to transp_out
                    float *out = g_out + (c_height-1)*c_rowstride + x;

                    int nmin = y-(WS-1);

#pragma unroll
                    for(;i>=nmin; --i, --bdata, out -= c_rowstride)
                    {
                        rev(**bdata*B0_2, e, c_weights);

                        if(x < c_width)
                            *out = e[0];
                    }
                }
                else
                {
                    float *out = g_out + y*c_rowstride + x;

#pragma unroll
                    for(int i=WS-1; i>=0; --i, --bdata, out -= c_rowstride)
                    {
                        rev(**bdata*B0_2, e, c_weights);

                        if(x < c_width)
                            *out = e[0];
                    }
                }
            }
        }
    }
}

